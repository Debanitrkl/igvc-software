#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <hiprand/hiprand_kernel.h>
#include "hip/hip_runtime.h"
#include "cuda_headers.h"
#include <hiprand.h>
#include <vector>

__global__ void kernel(int n, float *x, float *y) {
  int index = threadIdx.x;
  int stride = blockDim.x;
  for (int i = index; i < n; i += stride) {
    y[i] = x[i] + y[i];
  }
}

void cudamain() {

  dev_array

  hiprandGenerator_t curand_generator;
  hiprandCreateGenerator(&curand_generator, HIPRAND_RNG_PSEUDO_MTGP32);
  hiprandSetPseudoRandomGeneratorSeed(curand_generator, 1234ULL);
  hiprandGenerateNormal(curand_generator, d)


  kernel <<<1,256>>>(N, x, y);

  hipDeviceSynchronize();

  for (int i = 0; i < 10; i++) {
    std::cout << "x: " << x[i] << ", y: " << y[i] << std::endl;
  }

  hipFree(x);
  hipFree(y);
}
